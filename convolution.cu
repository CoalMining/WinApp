#include "hip/hip_runtime.h"

#include <iostream>

using namespace std;

int *firstMatrix, *secondMatrix;

int main(int argc, char* argv)
{
	int l1, m1;
	int l2, m2;

	cout << "Enter the dimension of the first matrix" << endl;
	cin >> l1 >> m1;

	cout << "Enter the dimension of the second matrix" << endl;
	cin >> l2 >> m2;

	cout << "Enter the elements of the first matrix\n\nRow major Order:" << endl;
	firstMatrix = new int[l1*m1];
	for (int i = 0; i < l1*m1; i++)
		cin >> firstMatrix[i];

	cout << "Enter the elements of the second matrix\n\nRow major Order:" << endl;
	secondMatrix = new int[l2*m2];
	for (int i = 0; i < l2*m2; i++)
		cin >> secondMatrix[i];


	delete[] firstMatrix;
	delete[] secondMatrix;
	return 0;
}