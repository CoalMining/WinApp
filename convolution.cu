#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <iostream>

using namespace std;


__global__ 
void convolution2D(int* firstMatrix, int* secondMatrix, int* resultMatrix, int l1, int m1, int l2, int m2);

int *firstMatrix, *kernelMatrix, *resultMatrix;

int main(int argc, char* argv)
{
	int l1, m1;
	int l2, m2;

	cout << "Enter the dimension of the first matrix" << endl;
	cin >> l1 >> m1;
	
	cout << "Enter the elements of the first matrix\n\nRow major Order:" << endl;
	firstMatrix = new int[l1*m1];
	resultMatrix = new int[l1*m1];
	for (int i = 0; i < l1*m1; i++)
		cin >> firstMatrix[i];

	cout << "Enter the dimension of the kernel matrix" << endl;
	cin >> l2 >> m2;

	cout << "Enter the elements of the kernel matrix\n\nRow major Order:" << endl;
	kernelMatrix = new int[l2*m2];
	for (int i = 0; i < l2*m2; i++)
		cin >> kernelMatrix[i];

	int *d_firstMatrix, *d_kernelMatrix, *d_resultMatrix;
	if (hipMalloc((void**)&d_firstMatrix, l1*m1 * sizeof(int)) != hipSuccess)
	{
		cout << "Error in allocating firstmatrix on device" << endl;
		return -1;
	}
	if (hipMalloc((void**)&d_resultMatrix, l1*m1 * sizeof(int)) != hipSuccess)
	{
		cout << "Error in allocating result matrix on device" << endl;
		return -1;
	}
	if (hipMalloc((void**)&d_kernelMatrix, l2*m2 * sizeof(int)) != hipSuccess)
	{
		cout << "Error in allocating kernel matrix on device" << endl;
		return -1;
	}

	if (hipMemcpy(d_firstMatrix, firstMatrix, sizeof(int)*l1*m1, hipMemcpyHostToDevice) != hipSuccess)
	{
		cout << "Error in copying data for first matrix to device" << endl;
		return -1;
	}

	if (hipMemcpy(d_kernelMatrix, kernelMatrix, sizeof(int)*l2*m2, hipMemcpyHostToDevice) != hipSuccess)
	{
		cout << "Error in copying data for second matrix to device" << endl;
		return -1;
	}

	//following section launches kernel
	convolution2D << < 1,dim3(m1,l1,1)>> > (d_firstMatrix,d_kernelMatrix, d_resultMatrix,l1,m1,l2,m2);
	if (hipDeviceSynchronize() != hipSuccess)
	{
		cout << "Error in synchronizing device" << endl;
		return -1;
	}

	if (hipMemcpy(resultMatrix,d_resultMatrix,sizeof(int)*l1*m1,hipMemcpyDeviceToHost)!=hipSuccess)
	{
		cout << "Error in copying data back from device to host" << endl;
		return -1;
	}

	cout << "The elements in row major order are as follows" << endl;
	for (int i = 0; i < l1*m1; i++)
	{
		cout << resultMatrix[i] << " ";
	}

	hipFree(d_firstMatrix);
	hipFree(d_kernelMatrix);
	hipFree(d_resultMatrix);

	delete[] firstMatrix;
	delete[] kernelMatrix;
	delete[] resultMatrix;


	return 0;
}


__global__ 
void convolution2D(int* firstMatrix, int* kernelMatrix, int* resultMatrix, int l1, int m1, int l2, int m2)
{
	//x is horizontal, y is vertical in Matrix
	//kernel call should be accordingly
	int tIdX = threadIdx.x + blockDim.x*blockIdx.x;
	int tIdY = threadIdx.y + blockDim.y*blockIdx.y;

	//l2 is no of rows so yShift comes from l2
	int yShift = l2 / 2;	//floor of l2/2
	int xShift = m2 / 2;	//floor of m2/2
	
	//starting index of element for the first matrix
	int yStart = tIdY - yShift;
	int xStart = tIdX - xShift;

	//private to each thread
	int tempRes = 0;

	for (int i = 0; i < l2 ; i++)
	{
		for (int j = 0; j < m2 ; j++)
		{
			//these loops are supposed to go through all the elememts of  the kernel matrix, so the loop

			//the element from the first matrix tthat are multiplied with the elements of the kernel matrix can be found by following:
			//	-> the X and Y index of the thread gives the center element
			//	-> subtract the xShift from X dim and yShift from Y dim to get the element of the first matrix to be multiplied with the first element of the kernel matrix
			//							(only if the index after shifting is greater than 0)
			//	-> since the shifting should go such that the elments of the kernel is mutliplied to corresponding element of first matrix add i and j
			if ((yStart + i) >= 0 && (xStart + j) >= 0 && (yStart + i)<l1 && (xStart + j)<m1)
			{
				tempRes += (kernelMatrix[i*m2 + j] * firstMatrix[(yStart + i)*m1 + (xStart + j)]);
			}
		}
	}
	resultMatrix[m1*tIdY + tIdX] = tempRes;
}